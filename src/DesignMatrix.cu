#include "hip/hip_runtime.h"
#include "DesignMatrix.hpp"
#include "omp.h"
#include <NumCpp.hpp>
#include <NumCpp/Functions/logical_or.hpp>
#include <NumCpp/Functions/where.hpp>
#include <cassert>
#include <cfloat>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <sys/types.h>

#define WARPSIZE 32

#define KERNEL(NAME, FIRST, SECOND)                                            \
  __global__ void NAME(size_t row_start, size_t row_end, size_t col_start,     \
                       size_t col_end, float *x, float *y, float *dataframe,   \
                       size_t *interaction, size_t *len_interact,              \
                       size_t *sample_idx, size_t df_ncol, size_t max_order) { \
                                                                               \
    __shared__ float partial_sums[WARPSIZE];                                   \
                                                                               \
    size_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;                 \
    size_t warp_idx = thread_idx / WARPSIZE;                                   \
    size_t lane_idx = thread_idx % WARPSIZE;                                   \
                                                                               \
    auto FIRST##_idx = FIRST##_start + warp_idx;                               \
                                                                               \
    if (FIRST##_idx < FIRST##_end) {                                           \
      partial_sums[lane_idx] = 0;                                              \
      for (int SECOND##_idx = SECOND##_start + lane_idx;                       \
           SECOND##_idx < SECOND##_end; SECOND##_idx += WARPSIZE) {            \
        size_t cur_sample_idx = sample_idx[col_idx];                           \
        size_t cur_len_interact = len_interact[col_idx];                       \
        bool nonzero = true;                                                   \
                                                                               \
        for (int j = 0; j < cur_len_interact; j++) {                           \
          size_t cur_interact = interaction[col_idx * max_order + j];          \
          float thres = dataframe[cur_sample_idx * df_ncol + cur_interact];    \
          float val = dataframe[row_idx * df_ncol + cur_interact];             \
          nonzero &= (val >= thres);                                           \
        }                                                                      \
                                                                               \
        if (nonzero) {                                                         \
          partial_sums[lane_idx] += x[SECOND##_idx - SECOND##_start];          \
        }                                                                      \
      }                                                                        \
    }                                                                          \
                                                                               \
    if (lane_idx == 0) {                                                       \
      float res = 0;                                                           \
      for (int lane_idx = 0; lane_idx < WARPSIZE; lane_idx++) {                \
        res += partial_sums[lane_idx];                                         \
      }                                                                        \
      y[FIRST##_idx - FIRST##_start] = res;                                    \
    }                                                                          \
  }

KERNEL(fusedRegionMV_kernel, row, col);
KERNEL(fusedRegionVM_kernel, col, row);

std::unique_ptr<nc::NdArray<float>>
DesignMatrix::fusedRegionMV(size_t row_start, size_t row_end, size_t col_start,
                            size_t col_end, const nc::NdArray<float> &x,
                            bool transpose) const {

  bool valid_row_bounds = (row_end <= this->_nrow) && (row_start <= row_end);
  bool valid_col_bounds = (col_end <= this->_ncol) && (col_start <= col_end);

  if (!valid_row_bounds || !valid_col_bounds) {
    throw std::out_of_range("Index out of range");
  }

  auto shifted_row_start = row_start + this->_offset;
  auto shifted_row_end = row_end + this->_offset;

  size_t res_len;
  if (!transpose) {
    res_len = row_end - row_start;
  } else {
    res_len = col_end - col_start;
  }

  auto res = std::make_unique<nc::NdArray<float>>(res_len, 1);

  float *x_cuda;
  float *y_cuda;
  auto size_x = x.shape().rows * sizeof(float);
  auto size_y = res_len * sizeof(float);
  hipMalloc(&x_cuda, size_x);
  hipMalloc(&y_cuda, size_y);
  hipMemcpy(x_cuda, x.data(), size_x, hipMemcpyHostToDevice);

  if (!transpose) {
    fusedRegionMV_kernel<<<res_len, WARPSIZE>>>(
        shifted_row_start, shifted_row_end, col_start, col_end, x_cuda, y_cuda,
        this->_dataframe_cuda, this->_interaction_cuda,
        this->_len_interact_cuda, this->_sample_idx_cuda,
        this->_dataframe.shape().cols, this->_max_order);
  } else {
    fusedRegionVM_kernel<<<res_len, WARPSIZE>>>(
        shifted_row_start, shifted_row_end, col_start, col_end, x_cuda, y_cuda,
        this->_dataframe_cuda, this->_interaction_cuda,
        this->_len_interact_cuda, this->_sample_idx_cuda,
        this->_dataframe.shape().cols, this->_max_order);
  }

  hipDeviceSynchronize();
  hipMemcpy(res->data(), y_cuda, res_len * sizeof(float),
             hipMemcpyDeviceToHost);
  hipFree(x_cuda);
  hipFree(y_cuda);

  return res;
}

void DesignMatrix::_init_ColIndices(size_t order, int prev_idx,
                                    std::vector<size_t> &interact) {
  if (order == 0) {
    auto num_sampled_row = this->_sampled_row.shape().cols;
    for (int i = 0; i < num_sampled_row; i++) {
      int row_idx = this->_sampled_row(0, i);
      this->ColIndices.push_back(
          ColIndex{interact, static_cast<size_t>(row_idx)});
    }
    return;
  }

  size_t df_ncol = this->_dataframe.shape().cols;
  for (int i = prev_idx + 1; i < df_ncol; i++) {
    interact.push_back(i);
    this->_init_ColIndices(order - 1, i, interact);
    interact.pop_back();
  }
}

DesignMatrix::DesignMatrix(const nc::NdArray<float> &dataframe,
                           size_t max_order, float sample_ratio, float reduce_epsilon)
    : _dataframe(dataframe), _max_order(max_order), _type("train"), _offset(0) {
  auto df_shape = dataframe.shape();
  size_t df_nrow = df_shape.rows;
  this->_nrow = df_nrow;

  if (sample_ratio == 1) {
    this->_sampled_row =
        nc::arange<int>(0, this->_nrow).reshape(1, this->_nrow);
  } else {
    uint32_t num_sampled_row = std::floor(this->_nrow * sample_ratio);
    this->_sampled_row =
        nc::random::randInt<int>({1, num_sampled_row}, 0, this->_nrow);
  }
  for (int o = 1; o <= max_order; o++) {
    std::vector<size_t> interact;
    this->_init_ColIndices(o, -1, interact);
  }

  this->_ncol = this->ColIndices.size();
  this->_allocate_cudamem();

  if (reduce_epsilon != -1) {
    this->reduce_basis(reduce_epsilon);
  }
};

DesignMatrix::DesignMatrix(const DesignMatrix &other) {
  this->_dataframe = other._dataframe;
  this->_type = other._type;
  this->_max_order = other._max_order;
  this->_nrow = other._nrow;
  this->_ncol = other._ncol;
  this->_offset = other._offset;
  this->_sampled_row = other._sampled_row;
  for (auto& c : other.ColIndices) {
    this->ColIndices.push_back({c.interaction, c.sample_idx});
  }
};

DesignMatrix::~DesignMatrix() {
  hipFree(_dataframe_cuda);
  hipFree(_interaction_cuda);
  hipFree(_len_interact_cuda);
  hipFree(_sample_idx_cuda);
}

void DesignMatrix::_init_PredDesignMatrix(const nc::NdArray<float> &new_df) {
  this->_offset = this->_dataframe.shape().rows;
  this->_dataframe = nc::stack({this->_dataframe, new_df}, nc::Axis::ROW);
  this->_type = "prediction";
  this->_nrow = new_df.shape().rows;
  this->_allocate_cudamem();
}

std::unique_ptr<DesignMatrix>
DesignMatrix::getPredDesignMatrix(const nc::NdArray<float> &new_df) const {
  if (this->_type != "train") {
    std::cerr << "Should be called from DesignMatrix with type `train`"
              << std::endl;
  }
  auto res = std::make_unique<DesignMatrix>(*this);
  res->_init_PredDesignMatrix(new_df);
  return res;
}

void DesignMatrix::_allocate_cudamem(bool reserve_df) {
  auto df_shape = this->_dataframe.shape();
  size_t df_nrow = df_shape.rows;
  size_t df_ncol = df_shape.cols;

  auto size_df = df_nrow * df_ncol * sizeof(float);
  auto size_interact = this->_ncol * this->_max_order * sizeof(size_t);
  auto size_len_interat = this->_ncol * sizeof(size_t);
  auto size_sample_idx = this->_ncol * sizeof(size_t);

  if (!reserve_df) {
    hipMalloc(&this->_dataframe_cuda, size_df);
  }
  hipMalloc(&this->_interaction_cuda, size_interact);
  hipMalloc(&this->_len_interact_cuda, size_len_interat);
  hipMalloc(&this->_sample_idx_cuda, size_sample_idx);

  size_t *arr_interaction = (size_t *)malloc(size_interact);
  size_t *arr_len_interact = (size_t *)malloc(size_len_interat);
  size_t *arr_sample_idx = (size_t *)malloc(size_len_interat);

  /* TODO: Can be parallelize */
  for (int i = 0; i < this->_ncol; i++) {
    auto &col_index = this->ColIndices[i];
    auto &interact = col_index.interaction;
    auto sample_idx = col_index.sample_idx;

    auto cur_len_interact = interact.size();
    arr_len_interact[i] = cur_len_interact;
    arr_sample_idx[i] = sample_idx;

    for (int j = 0; j < cur_len_interact; j++) {
      arr_interaction[i * this->_max_order + j] = interact[j];
    }
  }

  if (!reserve_df) {
    hipMemcpy(this->_dataframe_cuda, this->_dataframe.data(), size_df,
              hipMemcpyHostToDevice);
  }
  hipMemcpy(this->_interaction_cuda, arr_interaction, size_interact,
             hipMemcpyHostToDevice);
  hipMemcpy(this->_len_interact_cuda, arr_len_interact, size_len_interat,
             hipMemcpyHostToDevice);
  hipMemcpy(this->_sample_idx_cuda, arr_sample_idx, size_sample_idx,
             hipMemcpyHostToDevice);

  free(arr_interaction);
  free(arr_len_interact);
  free(arr_sample_idx);
}

std::unique_ptr<nc::NdArray<float>> DesignMatrix::proportion_ones() const {
  auto proportion_ones = this->fusedRegionMV(
    0,
    this->_nrow,
    0,
    this->_ncol,
    nc::ones<float>(this->_nrow, 1),
    true
  );
  (*proportion_ones) = (*proportion_ones) / (float)this->_nrow;

  return proportion_ones;
}

void DesignMatrix::reduce_basis(float epsilon) {
  assert(epsilon >= 0);

  auto proportion_ones = this->proportion_ones();

  #pragma omp parallel for
  for (int c = 0; c < proportion_ones->shape().rows; c++) {
    if ((*proportion_ones)(c, 0) == 0.0f) {
      (*proportion_ones)(c, 0) = 1.1;
    }
  }

  float lower_bound = nc::min(*proportion_ones)(0, 0) * (1 + epsilon);
  auto shouldRemoved = nc::logical_or((*proportion_ones) >= 1.0f,
                                      (*proportion_ones) < lower_bound);

  #pragma omp parallel for
  for (int c = 0; c < this->_ncol; c++) {
    if (shouldRemoved(c, 0)) {
      ColIndices[c]._to_be_removed = true;
    }
  }

  auto& ColIndices = this->ColIndices;
  ColIndices.erase(std::remove_if(ColIndices.begin(), ColIndices.end(),
                                  [&](const ColIndex& col) mutable {
                                    return col._to_be_removed;
                                  }),
                   ColIndices.end());

  this->_ncol = ColIndices.size();

  hipFree(this->_interaction_cuda);
  hipFree(this->_len_interact_cuda);
  hipFree(this->_sample_idx_cuda);
  
  this->_allocate_cudamem(true);
}


std::unique_ptr<nc::NdArray<bool>>
DesignMatrix::getCol(size_t col_idx, size_t start_idx, size_t end_idx) const {

  if (this->_type != "train") {
    std::cerr << "Should be called from DesignMatrix with type `train`"
              << std::endl;
  }
  const nc::NdArray<float> &df = this->_dataframe;
  auto& col_index = this->ColIndices[col_idx];

  // Check validality of col_index, return nullptr if invalid
  size_t interact_size = col_index.interaction.size();
  bool valid_sample_idx = (col_index.sample_idx < df.shape().rows);
  bool valid_interaction_size = (interact_size <= this->_max_order);
  bool increased = true;
  for (int i = 1; i < interact_size; i++) {
    increased &= (col_index.interaction[i - 1] < col_index.interaction[i]);
  }
  if (!valid_sample_idx | !valid_interaction_size | !increased) {
    return nullptr;
  }

  if ((start_idx == 0) & (end_idx == 0)) {
    end_idx = this->_nrow;
  }

  auto res = std::make_unique<nc::NdArray<bool>>(
      nc::ones<bool>(end_idx - start_idx, 1));
  for (auto c : col_index.interaction) {
    float thres = df(col_index.sample_idx, c);
    auto cur_col = df(nc::Slice(start_idx, end_idx), c);
    *res = nc::logical_and(*res, cur_col >= thres);
  }

  return res;
}

std::unique_ptr<BinSpMat> DesignMatrix::getRegion(uint64_t row_start,
                                                  uint64_t row_end,
                                                  uint64_t col_start,
                                                  uint64_t col_end) const {

  bool valid_row_bounds = (row_end <= this->_nrow) && (row_start <= row_end);
  bool valid_col_bounds = (col_end <= this->_ncol) && (col_start <= col_end);

  if (!valid_row_bounds || !valid_col_bounds) {
    throw std::out_of_range("Index out of range");
  }

  auto shifted_row_start = row_start + this->_offset;
  auto shifted_row_end = row_end + this->_offset;

  const size_t num_threads = omp_get_max_threads();
  const size_t row_size = shifted_row_end - shifted_row_start;
  const size_t col_size = col_end - col_start;
  size_t block_size = std::ceil(row_size / (float)num_threads);

  auto res = std::make_unique<BinSpMat>(row_size, col_size);

#pragma omp parallel
  {
    size_t thread_id = omp_get_thread_num();
    size_t block_row_start = shifted_row_start + thread_id * block_size;
    size_t block_row_end =
        shifted_row_start + std::min((thread_id + 1) * block_size, row_size);

    for (int row_idx = block_row_start; row_idx < block_row_end; row_idx++) {
      for (int col_idx = col_start; col_idx < col_end; col_idx++) {
        if (this->at(row_idx, col_idx)) {
          auto local_row_idx = row_idx - shifted_row_start;
          auto local_col_idx = col_idx - col_start;
          res->fill(local_row_idx, local_col_idx);
        }
      }
    }
  }

  res->translate();

  return res;
}

std::unique_ptr<BinSpMat> DesignMatrix::getBatch(const size_t start_idx,
                                                 const size_t end_idx) const {
  return getRegion(start_idx, end_idx, 0, this->_ncol);
}

bool DesignMatrix::at(const size_t row_idx, const size_t col_idx) const {
  auto &df = this->_dataframe;
  auto &col_index = this->ColIndices[col_idx];
  auto &interaction = col_index.interaction;
  auto sample_idx = col_index.sample_idx;

  bool res = true;
  for (auto &c : interaction) {
    float thres = df(sample_idx, c);
    float val = df(row_idx, c);
    res &= (val >= thres);
  }
  return res;
}
